
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <algorithm>
#include <vector>
#include <assert.h>

#define WIDTH 2048                      // Define the matrix width number
#define TILE_WIDTH 32                   // Define the tile width (same as thread block dimensions)
#define THREADS_PER_BLOCK_X TILE_WIDTH  // Define the number of threads in a block in x
#define THREADS_PER_BLOCK_Y TILE_WIDTH  // Define the number of threads in a block in y

inline hipError_t checkCuda(hipError_t result) {
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

// CUDA kernel to perform matrix multiplication
__global__ void matrixMultiplication(const float* M, const float* N, float* P, const int width) {
    // Declare shared matrices of size block*block (tiles)
    __shared__ float M_tile[TILE_WIDTH][TILE_WIDTH];
    __shared__ float N_tile[TILE_WIDTH][TILE_WIDTH];
    
    // For the sake of simplifying the notation, assign registers for thread_x,y
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Calculate the row and column index of the current element
    int row = blockIdx.y * TILE_WIDTH + ty;
    int col = blockIdx.x * TILE_WIDTH + tx;

    // Initialize the intermediate P value
    float sum = 0.;

    // Fill the shared memory
    //
    // Loop over the tiles of the input matrices
    for (int t = 0; t < width / TILE_WIDTH; ++t) {
        // Load (in collaboration with other threads) 
        // the tiles into shared memory
        if ( (row < width) && (t * TILE_WIDTH + tx < width) )
            M_tile[ty][tx] = M[row * width + t * TILE_WIDTH + tx];
        else 
            M_tile[ty][tx] = 0.;

        if ( (t * TILE_WIDTH + ty < width) && (col < width) )
            N_tile[ty][tx] = N[(t * TILE_WIDTH + ty) * width + col];
        else 
            N_tile[ty][tx] = 0.;

        // Synchronize (ensure the tile is loaded in shared memory)
        __syncthreads();
    
        // Perform the multiplication for this tile
        for (int k = 0; k < TILE_WIDTH; ++k) {
            sum += M_tile[ty][k] * N_tile[k][tx];
        }

        // Ensure all threads are done computing before loading the next tile
        __syncthreads(); 
    }

    // Write the result back to the global memory
    if (row < width && col < width) {
        P[row * width + col] = sum;
    }
}

// Function to generate a random number between 0 and 1
float random_number() {
    return (std::rand()*1./RAND_MAX);
}

// Function to printout the matrix
void print_matrix(const float* M, int rows, int cols) {
    if (WIDTH < rows) 
        rows = WIDTH;
    if (WIDTH < cols) 
        cols = WIDTH;
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%.2f", M[i * rows + j]);
            if (j < cols - 1) printf("\t");
        }
        printf("\n");
    }
}

int main(int argc, char** argv) {

    // Seed the random number generator with the current time
    srand(time(NULL));  // Ensure that rand() produces different sequences each run

    // Local vectors hosted in memory, each with N elements
    // using a vector to host the matrix, in a row-wise allocation
    std::vector<float> M(WIDTH * WIDTH), N(WIDTH * WIDTH), P(WIDTH * WIDTH);
    std::generate(M.begin(), M.end(), random_number); // Fill vector 'M' with random numbers
    std::generate(N.begin(), N.end(), random_number); // Fill vector 'N' with random numbers

    printf("Matrix M\n");
    print_matrix(M.data(), 10, 10);

    printf("Matrix N\n");
    print_matrix(N.data(), 10, 10);

    // Device matrices
    float* d_M;
    float* d_N;
    float* d_P;
    size_t matrixSize = WIDTH * WIDTH * sizeof(float);
    hipMalloc((void**)&d_M, matrixSize);
    hipMalloc((void**)&d_N, matrixSize);
    hipMalloc((void**)&d_P, matrixSize);

    // Copy host matrices to device
    hipMemcpy(d_M, M.data(), matrixSize, hipMemcpyHostToDevice);
    hipMemcpy(d_N, N.data(), matrixSize, hipMemcpyHostToDevice);

    // Compute the dimensions of blocks and grid
    // Blocks are now 2-dimensional
    dim3 blockSize(THREADS_PER_BLOCK_X,THREADS_PER_BLOCK_Y);
    dim3 gridSize(ceil(float(WIDTH)/blockSize.x),ceil(float(WIDTH)/blockSize.y));

    // Launch CUDA kernel
    matrixMultiplication<<<gridSize, blockSize>>>(d_M, d_N, d_P, WIDTH);

    // Copy the result vector from the GPU back to the CPU
    checkCuda(
        hipMemcpy(P.data(), d_P, matrixSize, hipMemcpyDeviceToHost)
    );

    printf("Matrix P\n");
    print_matrix(P.data(), 10, 10);

    // Cleanup by freeing the allocated GPU memory
    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_P);

    return 0;
}


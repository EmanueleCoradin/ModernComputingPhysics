
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <vector>     // For std::vector
#include <algorithm>  // For std::generate
#include <stdlib.h>   // For srand and rand
#include <time.h>     // For time

#define N 4  // Define the size of the vector

// Define a CUDA kernel that adds two vectors element-wise
__global__ void sum_kernel(const int *x, const int *y, int *res){
    // Calculate the thread ID
    int tid = threadIdx.x;
    printf("Thread number %d\n",tid);

    // Each thread computes one element of the result vector
    res[tid] = x[tid] + y[tid];
}

// Function to generate a random number between 0 and 99
int random_number() {
    return (std::rand() % 100);
}

int main(int argc, char **argv){

    // Seed the random number generator with the current time
    srand(time(NULL));  // Ensure that rand() produces different sequences each run

    // Local vectors hosted in memory, each with N elements
    std::vector<int> a(N), b(N), c(N);

    // Initialize vectors 'a' and 'b' with random numbers
    std::generate(a.begin(), a.end(), random_number);  // Fill vector 'a' with random numbers
    std::generate(b.begin(), b.end(), random_number);  // Fill vector 'b' with random numbers
    
    // Pointers to device (GPU) memory for the vectors
    int *dev_a, *dev_b, *dev_c;         

    // Determine the size of the memory required for each vector
    int size = N * sizeof(int);             

    // Allocate space on the GPU for the copies of the vectors
    hipMalloc((void **)&dev_a, size);
    hipMalloc((void **)&dev_b, size);
    hipMalloc((void **)&dev_c, size);

    // Print the result of vector addition on the CPU
    printf("CPU result:\n");
    for (int i = 0; i < N; i++) {
        printf("[el. %d] %d + %d = %d (on CPU) \n",i,a[i],b[i],a[i]+b[i]);
    }
    
    // Copy the input vectors from the CPU to the GPU
    hipMemcpy(dev_a, a.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b.data(), size, hipMemcpyHostToDevice);

    // Launch the sum kernel on the GPU with 1 block and N threads
    sum_kernel<<<1, N>>>(dev_a, dev_b, dev_c);
    
    // Copy the result vector from the GPU back to the CPU
    hipMemcpy(c.data(), dev_c, size, hipMemcpyDeviceToHost);    

    // Print the result of the vector addition performed on the GPU
    printf("GPU result:\n");
    for (int i = 0; i < N; i++) {
        printf("[el. %d] %d + %d = %d (on GPU) \n",i,a[i],b[i],c[i]);
    }
   
    // Cleanup by freeing the allocated GPU memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);        

    // Return 0 to indicate successful execution
    return 0;
}

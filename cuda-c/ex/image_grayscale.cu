
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// CUDA kernel to convert to grayscale
__global__ void rgb_to_grayscale(/* ... */) {    

    //...
}

// Function to read the PPM image into a 1D array
// (can be modified to read it into a 2D array or any other data types)
// 
int *read_pgm(const char *filename, int *width, int *height, int *max_val) {
    
    // Open the input file in read mode "r"
    FILE *file = fopen(filename, "r");
    
    // Check if file can be opened
    if (file == NULL) {
        printf("Could not open file.\n");
        return NULL;
    }

    // Read the PPM header, composed by 3 lines, e.g.:
    //
    // P3                           [magic number]
    // 1024 768                     [pixel_width pixel_height]
    // 65535                        [max color levels]
    //
    // More info here -- https://www.wikiwand.com/en/articles/Netpbm

    // Read the first line, and verify if it states `P3` 
    char format[3];
    fscanf(file, "%s", format);
    if (format[0] != 'P' || format[1] != '3') {
        printf("Not a valid PGM (ASCII P3) file.\n");
        fclose(file);
        return NULL;
    }

    // Read the width, height, and maximum grayscale value
    fscanf(file, "%d %d", width, height);
    fscanf(file, "%d", max_val);

    // Compute the total amount of pixels
    int total_pixels = (*width) * (*height);

    // Allocate host memory for the image data
    // 3 x the image size to allocate R G B values
    int *image = (int *)malloc(3 * total_pixels * sizeof(int));
    
    // Read pixel values into the array
    // R0 G0 B0 R1 G1 B1 ...
    for (int i = 0; i < total_pixels * 3; i++) {
        fscanf(file, "%d", &image[i]);
    }

    // Close the input file
    fclose(file);  

    // Return the pixel array
    return image;  
}

// Function to write the PGM image from a 1D array
void write_pgm(const char *filename, int *image, int width, int height, int max_val) {

    // Open the output file in write mode "w"
    FILE *file = fopen(filename, "w");

    // Check if file can be opened
    if (file == NULL) {
        printf("Could not open file for writing.\n");
        return;
    }

    // Write the PGM header
    fprintf(file, "P2\n");
    fprintf(file, "%d %d\n", width, height);
    fprintf(file, "%d\n", max_val);

    // Write the pixel values
    for (int i = 0; i < width * height; i++) {
        fprintf(file, "%d ", image[i]);
        // Include a newline every "width" number of pixels
        if ((i + 1) % width == 0) {
            fprintf(file, "\n");
        }
    }

    // Close the output file
    fclose(file);  
}

int main() {
    int width, height, max_val;

    // Read the PGM image
    int *host_rgb = read_pgm("ny.ppm", &width, &height, &max_val);
    if (host_rgb == NULL) {
        return 1;  // Error reading the file
    }

    // Allocate memory for the grayscale image on the host

    // Allocate memory for the RGB image and Grayscale image on the GPU

    // Copy the RGB image data from the host to the device (GPU)
    
    // Define the block and grid dimensions
    
    // Launch the CUDA kernel to convert RGB to Grayscale

    // Copy the Grayscale image data back to the host
    
    // Write the grayscale image to a new PGM file
    
    // Free the memory on the host and the GPU

    return 0;
}


#include <hip/hip_runtime.h>
#include <stdio.h>

// Define a CUDA kernel that adds two integers
__global__ void sum_kernel(int *x, int *y, int *res){
    // All operands are passed by reference
    //
    // The operation is executed on the device,
    // so the variables x, y, res must point to GPU memory
    
    *res = *x + *y;  // Perform addition on the GPU
}

int main(int argc, char **argv){

    // Local variables, hosted in the CPU memory
    int a, b, c;                        
    
    // Pointers to device (GPU) memory for the variables
    int *dev_a, *dev_b, *dev_c;         

    // Determine the size of the memory required for each integer
    int size = sizeof(int);             

    // Allocate space on the GPU for the copies of the variables (both inputs and output)
    hipMalloc((void **)&dev_a, size);
    hipMalloc((void **)&dev_b, size);
    hipMalloc((void **)&dev_c, size);

    // Initialize the input variables on the CPU
    a = 2;
    b = 3;

    // Print the result of addition on the CPU
    printf("%d + %d = %d (on CPU) \n", a, b, a + b);
    
    // Copy the input variables from the CPU to the GPU
    hipMemcpy(dev_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, &b, size, hipMemcpyHostToDevice);

    // Launch the sum kernel on the GPU with 1 block and 1 thread (non-blocking operation)
    sum_kernel<<<1,1>>>(dev_a, dev_b, dev_c);
    
    // Copy the result from the GPU back to the CPU (blocking operation)
    hipMemcpy(&c, dev_c, size, hipMemcpyDeviceToHost);    

    // Print the result of the addition performed on the GPU
    printf("%d + %d = %d (on GPU) \n", a, b, c);
   
    // Cleanup by freeing the allocated GPU memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);        

    return 0;

}
